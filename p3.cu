#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stddef.h>
#include<search.h>
#include<hip/device_functions.h>
#define HANDLE_ERROR( err ) {if (err != hipSuccess) { \
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), __FILE__, __LINE__ ); \
        exit( EXIT_FAILURE );}}
#include "util/cuPrintf.cu"
#define MAX_FILE_SIZE 200
#define MAX_HASH_ENTRIES 200
#define M 100
#define BAGSIZE 10

__global__ void getWordCounts(char *fileArray,int *countArray,int *fileSize,char *wordhashtable, int *nextPtr, int *lock){
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  int ind,word_started =0 ,count =0;
  int found;
  int hashvalue;
  char *ptr,*wptr,*temp;
  ptr = &fileArray[i*MAX_FILE_SIZE];
  int  tempi=0,tempi2;


  for(ind =0;ind<fileSize[i];ind++){
//    if(ptr[ind]>64&&ptr[ind]<91) ptr[ind]+=32;   //ADD FOR CASE INSENSITIVE
    if(ptr[ind]!=' '&&ptr[ind]!='.'&&ptr[ind]!='!'&&ptr[ind]!='\n')
      if(word_started!=1) {
	word_started = 1;
	hashvalue = ptr[ind];//>64&&ptr[ind]<91) ? ptr[ind]+32:ptr[ind];//temp addition else do only assignemnt
	wptr = &ptr[ind];
      }
      else{//Middle of the word
	hashvalue+= ptr[ind];//>64&&ptr[ind]<91) ? ptr[ind]+32:ptr[ind];
      }
    if(word_started)
      if(ptr[ind]==' '||ptr[ind]=='.'||ptr[ind]=='!'||ptr[ind]=='\n'){
        word_started = 0;
	hashvalue = hashvalue % M;// 10 here is hashtable size M
	//cuPrintf("\n%d : Found token %c%c%c",i,wptr[0],wptr[1],wptr[2]);

	/*CHECK IF FOUND IN HASH
	*---------------------------
	*/	
	//lock -hashvalue
//	cuPrintf("\n%d : 1locking  %d",i,hashvalue);
	while(!atomicCAS(&lock[hashvalue],0,1));
	if(wordhashtable[hashvalue*20]=='\0'){//Not found in Hash

	  temp = &wordhashtable[hashvalue*20];
	  tempi =0;
	  while(&wptr[tempi]!=&ptr[ind])//Entering in hash table
		{temp[tempi]= wptr[tempi];
		tempi++;}
	//unlock -hash value
	//  if(!atomicCAS(&lock[hashvalue],1,0)) atomicAdd(&countArray[hashvalue],999);//error detection
//	cuPrintf("\n%d : 1UNlocking  %d",i,hashvalue);
	atomicExch(&lock[hashvalue],0);
	  atomicAdd(&countArray[hashvalue],1);//count

	}
	else{

	  /*CHECK IF COLLISION
	   *---------------------*/
	  tempi =hashvalue;found = -1;
	 /*Check word*/
	  while(nextPtr[tempi]!=-1||found==-1){
//	    cuPrintf("\n%d: next check : %d",i,tempi);
	    tempi2 = 0;
	    found =1;
	    temp = &wordhashtable[tempi*20];
//	    cuPrintf("\n%d: with word : %c%c%c",i,temp[0],temp[1],temp[2]);
	    while(&wptr[tempi2]!=&ptr[ind]){
	      if(temp[tempi2]!=wptr[tempi2]) {found =0;break;}
	      tempi2++;
	    }
	    if(temp[tempi2]!='\0') found =0;
	    if(found) break;
	    if(nextPtr[tempi]!=-1){
//		cuPrintf("\n%d : 2going to next link   %d",i,nextPtr[tempi]);
//		cuPrintf("\n%d : 2Unlocking  %d",i,tempi);found =-1;
		found =-1;
	    	//unlock - tempi
	    	atomicExch(&lock[tempi],0);
	       	tempi = nextPtr[tempi];      
		//lock - tempi
//		cuPrintf("\n%d : 2locking  %d",i,tempi);
		while(!atomicCAS(&lock[tempi],0,1));
	    }
	  }

	  if(found){ /*COLLISION BUT FOUND*/
	    //unlock - tempi
//		cuPrintf("\n%d : 3Unlocking  %d",i,tempi);
	    atomicExch(&lock[tempi],0);
	    atomicAdd(&countArray[tempi],1);
		}

	  else{   /*COLLISION AND ENTRY NOT FOUND*/
	    tempi2 =0;
	    //lock - M+tempi2
//		cuPrintf("\n%d : 3locking  %d",i,M+tempi2);
	    while(!atomicCAS(&lock[M+tempi2],0,1));
	    while(wordhashtable[(M+tempi2)*20]!='\0' && (tempi2+M)<MAX_HASH_ENTRIES) {
//		cuPrintf("\n%d : 4Unlocking  %d",i,M+tempi2);
		//unlock - M+tempi2
	        atomicExch(&lock[M+tempi2],0);
		tempi2++;//10 = M; tempi2 holds location in hast tab;e
		//lock - M+tempi2
//		cuPrintf("\n%d : 4locking  %d",i,M+tempi2);
	    	while(!atomicCAS(&lock[M+tempi2],0,1));
	    }
	    if((tempi2+M) < MAX_HASH_ENTRIES){
//		cuPrintf("\n%d : updating nextPtr at %d to %d",i,tempi,M+tempi2);
	    	nextPtr[tempi] = tempi2+M;
	    	//unlock - tempi
//		cuPrintf("\n%d : 5Unlocking  %d",i,tempi);
	    	atomicExch(&lock[tempi],0);
		
		tempi=0;//tempi holds the location where last hash was found not needed anymore.
	        temp = &wordhashtable[(M+tempi2)*20];
		while(&wptr[tempi]!=&ptr[ind]) //Entering in hash table
			{temp[tempi]= wptr[tempi]; 
			tempi++;}

		atomicAdd(&countArray[tempi2+M],1);
	    }//count*/
	   //unlock - tempi
//		cuPrintf("\n%d : 6Unlocking  %d",i,tempi);
	    atomicExch(&lock[tempi],0);//succeeds when there was no space in hash
	    //tryunlock = M+tempi2
//		cuPrintf("\n%d : 7UNlocking  %d",i,M+tempi2);
	    atomicExch(&lock[M+tempi2],0);
	  }

	}//else of collision detection
	count++;
      }//if ptr = delimiter...
  }//for
}

int main(int argc,char **argv){
  char *filename=NULL;//Limiting no if files
  char *fileArray;
  char *dfileArray;
  char *dfileArray1;
  int *countArray;
  int *dcountArray;
  int *fileSize;
  int *dfileSize;
  int *dfileSize1;
  char *hashtable; 
  char *dhashtable; 
  int *nextPtr;
  int *dnextPtr;
  int *dlock;
  int noOfFiles=0;
  FILE *fp;
  char c;
  hipStream_t stream,stream2;
  HANDLE_ERROR( hipStreamCreate(&stream) );
  HANDLE_ERROR( hipStreamCreate(&stream2) );

  int readComplete =1 ,noOfReads=0 , redArea =0 ;

 // initialize cuPrintf
  cudaPrintfInit();


  char *temp;int itemp=0,itemp2 =0 ,max,maxind;
  filename =(char*) malloc (10*sizeof(char));
  //fileArray=(char*) malloc(10*MAX_FILE_SIZE*sizeof(char));
  hipHostAlloc((void**)&fileArray,10*MAX_FILE_SIZE*sizeof(char),hipHostMallocDefault);
 // hipHostMalloc((void**)&fileArray,10*MAX_FILE_SIZE*sizeof(char));
  countArray =(int*) malloc (MAX_HASH_ENTRIES*sizeof(int));//corresponding counts of words
  //fileSize =(int*) malloc (10*sizeof(int));
  hipHostAlloc((void**)&fileSize,10*sizeof(int),hipHostMallocDefault);
  //hipHostMalloc((void**)&fileSize,10*sizeof(int));
  hashtable=(char*) malloc(20*MAX_HASH_ENTRIES*sizeof(char));
  nextPtr = (int*) malloc (MAX_HASH_ENTRIES*sizeof(int));

  hipMalloc((void**)&dfileArray,10*MAX_FILE_SIZE*sizeof(char));
  hipMalloc((void**)&dfileArray1,10*MAX_FILE_SIZE*sizeof(char));
  hipMalloc((void**)&dcountArray,MAX_HASH_ENTRIES*sizeof(int));//corresponding counts of words
  hipMalloc((void**)&dfileSize,10*sizeof(int));
  hipMalloc((void**)&dfileSize1,10*sizeof(int));
  hipMalloc((void**)&dhashtable,20*MAX_HASH_ENTRIES*sizeof(char));//20-max word size 500-max words
  hipMalloc((void**)&dnextPtr,MAX_HASH_ENTRIES*sizeof(int));//corresponding counts of words
  hipMalloc((void**)&dlock,MAX_HASH_ENTRIES*sizeof(int));//corresponding counts of words

  hipMemset(dcountArray,0,MAX_HASH_ENTRIES*sizeof(int));
  hipMemset(dhashtable,'\0',20*MAX_HASH_ENTRIES*sizeof(char));
  hipMemset(dnextPtr,-1,MAX_HASH_ENTRIES*sizeof(int));
  hipMemset(dlock,0,MAX_HASH_ENTRIES*sizeof(int));

while(1)  {
 noOfFiles=0; 
  while(noOfReads<BAGSIZE&&(!readComplete || scanf("%s",filename)!=EOF)){
  //  printf("\nAttempting to open %s %d",filename,readComplete);
    if(readComplete){
      fp = fopen(filename,"r");
      if(fp == NULL) {
	        perror("failed to open sample.txt");
        	exit(0) ;//EXIT_FAILURE;
      }
    }
    memset(&fileArray[noOfFiles*200],'\0',MAX_FILE_SIZE);
    itemp = fread(&fileArray[noOfFiles*200],sizeof(char),MAX_FILE_SIZE,fp);
    //printf("\n->%s\n",&fileArray[noOfFiles*200]);
    if(feof(fp)) readComplete = 1;
    else readComplete =0 ;


//printf("\nGonna check edge case:(readComplete=%d)\n",readComplete);
    if(!readComplete){//handle edge
	c = fileArray[(noOfFiles*200)+199];itemp2 =199 ;
//			printf("noOfFiles=%d\n%c%c--",noOfFiles,c,fileArray[(noOfFiles*200)+198]);
	if(c>64 && c<91 || c>96 && c<123){
		while(c!=' '&&c!='.'&&c!='!'&&c!='\n'){
//			printf("%c",c);
		  fileArray[(noOfFiles*200)+itemp2]=' ';
		  itemp2--;
		  c = fileArray[(noOfFiles*200)+itemp2];
		}
//		printf("moving back %d\n",itemp2-200);
		fseek(fp,itemp2-200,SEEK_CUR);
	}
    }

    fileSize[noOfFiles]=itemp;//(itemp == 1) ? MAX_FILE_SIZE:ftell(fp)%MAX_FILE_SIZE;
		//printf("FileSize : %d\n",fileSize[noOfFiles]);
    if(readComplete){
      fclose(fp);fp = NULL;}
      noOfFiles++;
    
    noOfReads++;
  }

  //DEBUG: prints file
  temp = fileArray;itemp =0;
  while(itemp<noOfFiles){
//    printf("\n%s\n",temp);
	itemp++;
    temp+=200;
  }

  hipMemset(dlock,0,MAX_HASH_ENTRIES*sizeof(int));
  if(redArea == 0){
  HANDLE_ERROR( hipMemcpyAsync(dfileArray,fileArray,10*MAX_FILE_SIZE*sizeof(char),hipMemcpyHostToDevice,stream) );
  HANDLE_ERROR( hipMemcpyAsync(dfileSize, fileSize, 10*sizeof(int),               hipMemcpyHostToDevice,stream) );
  HANDLE_ERROR( hipStreamSynchronize( stream ) );
  getWordCounts<<<1,noOfFiles,0,stream>>>(dfileArray,dcountArray,dfileSize,dhashtable,dnextPtr, dlock);}
  else{
  HANDLE_ERROR( hipMemcpyAsync(dfileArray1,fileArray,10*MAX_FILE_SIZE*sizeof(char),hipMemcpyHostToDevice,stream2) );
  HANDLE_ERROR( hipMemcpyAsync(dfileSize1,fileSize,10*sizeof(int),hipMemcpyHostToDevice,stream2) );
  HANDLE_ERROR( hipStreamSynchronize( stream2 ) );
  getWordCounts<<<1,noOfFiles,0,stream2>>>(dfileArray1,dcountArray,dfileSize1,dhashtable,dnextPtr, dlock);
  }

  redArea = (redArea==0) ? 1:0;
  if(noOfReads == BAGSIZE) noOfReads =0 ;
  else break;
}//while (1)Q
  hipStreamSynchronize(stream);
  hipStreamSynchronize(stream2);
  hipDeviceSynchronize();
  hipMemcpy(countArray,dcountArray,200*sizeof(int),hipMemcpyDeviceToHost);
  hipMemcpy(hashtable,dhashtable,20*200*sizeof(char),hipMemcpyDeviceToHost);
 // launch a kernel with a single thread to greet from the device
  cudaPrintfDisplay();

  // clean up after cuPrintf
  cudaPrintfEnd();


  itemp=itemp2=0;max=-99,maxind = 0;
//  printf("\nNo Of Words : \n");
  for(itemp=0;itemp<200;itemp++){
	for(itemp2=0;itemp2<200;itemp2++)
	    if(hashtable[itemp2*20]!='\0'&&countArray[itemp2]!=0)
		if(countArray[itemp2] > max){
		  max = countArray[itemp2];
		  maxind = itemp2;
		}
	if(max!=-99){
      printf("%s:[%d]\n",&hashtable[maxind*20],max);
	max =-99; countArray[maxind] =0;
	}
	else break;
  }
		
  itemp =0;
/*  while(itemp<200){
//    printf("\t%d",countArray[itemp]);itemp++;
    if(hashtable[itemp*20]!='\0'){
      printf("%d: %s:[%d]\n",itemp,&hashtable[itemp*20],countArray[itemp]);
    }
    itemp++;
  }*/
  hipFree(dfileArray);
  hipFree(dfileArray1);
  hipFree(dcountArray);
  hipFree(dhashtable);
  hipHostFree(fileArray);
  hipHostFree(fileSize);
  free(countArray);
  free(hashtable);
}
